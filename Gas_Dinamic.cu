#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <cfloat>
#include <fstream>
#include <math.h>
#include <vector>
#include <string>
#include "Header.h"

using namespace std;

//__device__ int sign_(const double& x);
//__device__ double minmod_(double x, double y);
//__device__ double linear_(double x1, double t1, double x2, double t2, double x3, double t3, double y);
//__device__ void linear2_(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
//    double& A, double& B);



__device__ double minmod_(double x, double y)
{
    if (sign_(x) + sign_(y) == 0)
    {
        return 0.0;
    }
    else
    {
        return   ((sign_(x) + sign_(y)) / 2.0) * min(fabs(x), fabs(y));  ///minmod
        //return (2*x*y)/(x + y);   /// vanleer
    }
}

__device__ double linear_(double x1, double t1, double x2, double t2, double x3, double t3, double y)
{
    double d = minmod_((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    return  (d * (y - x2) + t2);
}

__device__ void linear2_(double x1, double t1, double x2, double t2, double x3, double t3, double y1, double y2,//
    double& A, double& B)
{
    // ������� �������� - ����������� - �� ������� �� ����
    double d = minmod_((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    A = (d * (y1 - x2) + t2);
    B = (d * (y2 - x2) + t2);
    //printf("%lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf | %lf \n", x1, t1, x2, t2, x3, t3, y1, y2, A, B);
    return;
}

__device__ int sign_(const double& x)
{
    if (x > 0)
    {
        return 1;
    }
    else if (x < 0)
    {
        return  -1;
    }
    else
    {
        return 0;
    }
}


__device__ double POTOK_Korolkov(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L,  double* P, const double& n1, const double& n2, const double& n3)
{
    double bx_L = Bx_L / spi4;
    double by_L = By_L / spi4;
    double bz_L = Bz_L / spi4;

    double t1 = 0.0;
    double t2 = 0.0;
    double t3 = 0.0;

    double m1 = 0.0;
    double m2 = 0.0;
    double m3 = 0.0;

    if (n1 > 0.1)
    {
        t2 = 1.0;
        m3 = 1.0;
    }
    else if (n2 > 0.1)
    {
        t3 = 1.0;
        m1 = 1.0;
    }
    else if (n3 > 0.1)
    {
        t1 = 1.0;
        m2 = 1.0;
    }
    else if (n1 < -0.1)
    {
        t3 = -1.0;
        m2 = -1.0;
    }
    else if (n2 < -0.1)
    {
        t1 = -1.0;
        m3 = -1.0;
    }
    else if (n3 < -0.1)
    {
        t1 = -1.0;
        m2 = -1.0;
    }
    else
    {
        printf("EROROR 1421  normal_error\n");
    }


    double u1, v1, w1, u2, v2, w2;
    u1 = v1_L * n1 + v2_L * n2 + v3_L * n3;
    v1 = v1_L * t1 + v2_L * t2 + v3_L * t3;
    w1 = v1_L * m1 + v2_L * m2 + v3_L * m3;

    double bn1, bt1, bm1, bn2, bt2, bm2;
    bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
    bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
    bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;

    //cout << " = " << bt2 * bt2 + bm2 * bm2 << endl;


    double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);

    double uu_L = (kv(v1_L) + kv(v2_L) + kv(v3_L)) / 2.0;


    double pTL = p_L + bb_L / 2.0;


    double FL[9];

    double e1 = p_L / g1 + ro_L * uu_L + bb_L / 2.0;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + pTL - kv(bn1);
    FL[2] = ro_L * u1 * v1 - bn1 * bt1;
    FL[3] = ro_L * u1 * w1 - bn1 * bm1;
    FL[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
    //cout << uu_L << endl;
    FL[5] = 0.0;
    FL[6] = u1 * bt1 - v1 * bn1;
    FL[7] = u1 * bm1 - w1 * bn1;
    FL[8] = Q_L * u1;


    double  PO[9];
        
        for (int i = 0; i < 9; i++)
        {
            PO[i] = FL[i];
        }

    P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
    P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
    P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
    P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
    P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
    P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
    P[0] = PO[0];
    P[4] = PO[4];

    double SWAP = P[4];
    P[4] = P[5];
    P[5] = P[6];
    P[6] = P[7];
    P[7] = SWAP;
    return;
}


__device__ void TVD(const double2& s_1, const double2& s_2, const double2& s_3, const double2& s_4, const double2& s_5,//
    const double2& s_6, const double2& s_7, const double2& s_8, const double2& s_9, double2& s12,//
    double2& s13, double2& s14, double2& s15, double2& s21, double2& s31, double2& s41, double2& s51, double dx, double dy, bool zero)
{
    // ��� ��������� � �������� zero ������ ���� ����� true
    linear2_(-dx, s_4.x, 0.0, s_1.x, dx, s_2.x, -dx / 2.0, dx / 2.0, s14.x, s12.x);
    if (zero == true)
    {
        if (s14.x <= 0.0)
        {
            s14.x = s_1.x;
        }
        if (s12.x <= 0.0)
        {
            s12.x = s_1.x;
        }
    }
   
    linear2_(-dx, s_4.y, 0.0, s_1.y, dx, s_2.y, -dx / 2.0, dx / 2.0, s14.y, s12.y);
    if (zero == true)
    {
        if (s14.y <= 0.0)
        {
            s14.y = s_1.y;
        }
        if (s12.y <= 0.0)
        {
            s12.y = s_1.y;
        }
    }

    linear2_(-dy, s_3.x, 0.0, s_1.x, dy, s_5.x, -dy / 2.0, dy / 2.0, s13.x, s15.x);
    if (zero == true)
    {
        if (s13.x <= 0.0)
        {
            s13.x = s_1.x;
        }
        if (s15.x <= 0.0)
        {
            s15.x = s_1.x;
        }
    }

    linear2_(-dy, s_3.y, 0.0, s_1.y, dx, s_5.y, -dy / 2.0, dy / 2.0, s13.y, s15.y);
    if (zero == true)
    {
        if (s13.y <= 0.0)
        {
            s13.y = s_1.y;
        }
        if (s15.y <= 0.0)
        {
            s15.y = s_1.y;
        }
    }

    s21.x = linear_(0.0, s_1.x, dx, s_2.x, 2.0 * dx, s_6.x, dx / 2.0);
    if (s21.x <= 0) s21.x = s_2.x;
    s21.y = linear_(0.0, s_1.y, dx, s_2.y, 2.0 * dx, s_6.y, dx / 2.0);
    if (s21.y <= 0) s21.y = s_2.y;

    s41.x = linear_(0.0, s_1.x, - dx, s_4.x, - 2.0 * dx, s_8.x, - dx / 2.0);
    if (s41.x <= 0 && zero == true) s41.x = s_4.x;
    s41.y = linear_(0.0, s_1.y, - dx, s_4.y, - 2.0 * dx, s_8.y, - dx / 2.0);
    if (s41.y <= 0 && zero == true) s41.y = s_4.y;

    s31.x = linear_(0.0, s_1.x, - dy, s_3.x, - 2.0 * dy, s_7.x, - dy / 2.0);
    if (s31.x <= 0 && zero == true) s31.x = s_3.x;
    s31.y = linear_(0.0, s_1.y, - dy, s_3.y, - 2.0 * dy, s_7.y, - dy / 2.0);
    if (s31.y <= 0 && zero == true) s31.y = s_3.y;

    s51.x = linear_(0.0, s_1.x, + dy, s_5.x, + 2.0 * dy, s_9.x, + dy / 2.0);
    if (s51.x <= 0) s51.x = s_5.x;
    s51.y = linear_(0.0, s_1.y, + dy, s_5.y, + 2.0 * dy, s_9.y, + dy / 2.0);
    if (s51.y <= 0)  s51.y = s_5.y;

    return;
}


__device__ double HLLC_Korolkov_2D(const double2& Ls, const double2& Lu, const double2& Rs, const double2& Ru,//
    const double n1, const double n2, double2& Ps, double2& Pu, const double rad)
{
    double u_L, v_L;
    double u_R, v_R;

    double ro1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double p1 = Ls.y;

    double ro2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double p2 = Rs.y;

    double t1 = -n2;    // ����������� ������
    double t2 = n1;

    u_L = u1 * n1 + v1 * n2;
    v_L = u1 * t1 + v1 * t2;

    u_R = u2 * n1 + v2 * n2;
    v_R = u2 * t1 + v2 * t2;

    double cL = sqrt(ga * p1 / ro1);
    double cR = sqrt(ga * p2 / ro2);

    double SL = min((u_L - cL), (u_R - cR));
    double SR = max((u_L + cL), (u_R + cR));

   /* double SL = min(u_L, u_R) - max(cL, cR);
    double SR = max(u_L, u_R) + max(cL, cR);*/

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    if (SL >= 0.0)
    {
        Ps.x = ro1 * u_L;
        Ps.y = ( ga * p1/(g1) + 0.5 * ro1 * (kv(u1) + kv(v1)) ) * u_L;
        Pu.x = (ro1 * u_L * u_L + p1) * n1 + (ro1 * u_L * v_L) * t1;
        Pu.y = (ro1 * u_L * u_L + p1) * n2 + (ro1 * u_L * v_L) * t2;
        return time;
    }
    else if (SR <= 0.0)
    {
        Ps.x = ro2 * u_R;
        Ps.y = (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;
        Pu.x = (ro2 * u_R * u_R + p2) * n1 + (ro2 * u_R * v_R) * t1;
        Pu.y = (ro2 * u_R * u_R + p2) * n2 + (ro2 * u_R * v_R) * t2;
        return time;
    }
    else
    {
        double SM = ( (SR - u_R)*ro2 * u_R - (SL - u_L)*ro1*u_L - p2 + p1 )/( (SR - u_R)*ro2 - (SL - u_L)*ro1 );
        double pp = p1 + ro1 * (SL - u_L) * (SM - u_L);

        if (SM <= 0.0)
        {
            double rr = ro2 * (SR - u_R) / (SR - SM);
            double e = p2 / g1 + 0.5 * ro2 * (kv(u2) + kv(v2));
            double ee = ((SR - u_R) * e - p2 * u_R + pp * SM) / (SR - SM);
            Ps.x = SR * (rr - ro2) + ro2 * u_R;
            Ps.y = SR * (ee - e) + (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;

            double F1 = (ro2 * u_R * u_R + p2) + SR * (rr * SM - ro2 * u_R);
            double F2 = (ro2 * u_R * v_R) + SR * (rr * v_R - ro2 * v_R);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else if (SM >= 0.0)
        {
            double rr = ro1 * (SL - u_L) / (SL - SM);
            double e = p1 / g1 + 0.5 * ro1 * (kv(u1) + kv(v1));
            double ee = ( (SL - u_L)*e - p1*u_L + pp * SM )/(SL - SM);
            Ps.x = SL * (rr - ro1) + ro1 * u_L;
            Ps.y = SL * (ee - e) + (ga * p1 / (g1) + 0.5 * ro1 * (kv(u1) + kv(v1))) * u_L;
            double F1 = (ro1 * u_L * u_L + p1) + SL * (rr * SM - ro1 * u_L);
            double F2 = (ro1 * u_L * v_L ) + SL * (rr * v_L - ro1 * v_L);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else
        {
            printf("ERROR HLLC_KOROLKOV_2d   kod oshibki: 1jdt27453h\n");
            return time;
        }
    }
    return time;
}

__device__ double HLLCQ_Korolkov_2D(const double2& Ls, const double2& Lu, const double2& Rs, const double2& Ru,//
    const double& LQ, const double& RQ, double n1, double n2, double2& Ps, double2& Pu, double& PQ, double rad)
{
    double u_L, v_L;
    double u_R, v_R;

    double ro1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double p1 = Ls.y;
    double Q_L = LQ;

    double ro2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double p2 = Rs.y;
    double Q_R = RQ;

    double t1 = -n2;    // ����������� ������
    double t2 = n1;

    u_L = u1 * n1 + v1 * n2;
    v_L = u1 * t1 + v1 * t2;

    u_R = u2 * n1 + v2 * n2;
    v_R = u2 * t1 + v2 * t2;

    double cL = sqrt(ga * p1 / ro1);
    double cR = sqrt(ga * p2 / ro2);

    double SL = min((u_L - cL), (u_R - cR));
    double SR = max((u_L + cL), (u_R + cR));

   /* double SL = min(u_L, u_R) - max(cL, cR);
    double SR = max(u_L, u_R) + max(cL, cR);*/

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double FL1 = ro1 * u_L * u_L + p1;
    double FL2 = ro1 * u_L * v_L;

    double FR1 = ro2 * u_R * u_R + p2;
    double FR2 = ro2 * u_R * v_R;

    if (SL >= 0.0)
    {
        PQ = Q_L * u_L;
        Ps.x = ro1 * u_L;
        Ps.y = (ga * p1 / (g1) + 0.5 * ro1 * (kv(u1) + kv(v1))) * u_L;
        Pu.x = (FL1) * n1 + (FL2) * t1;
        Pu.y = (FL1) * n2 + (FL2) * t2;
        return time;
    }
    else if (SR <= 0.0)
    {
        PQ = Q_R * u_R;
        Ps.x = ro2 * u_R;
        Ps.y = (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;
        Pu.x = (FR1) * n1 + (FR2) * t1;
        Pu.y = (FR1) * n2 + (FR2) * t2;
        return time;
    }
    else
    {
        double SM = ((SR - u_R) * ro2 * u_R - (SL - u_L) * ro1 * u_L - p2 + p1) / ((SR - u_R) * ro2 - (SL - u_L) * ro1);
        double pp = p1 + ro1 * (SL - u_L) * (SM - u_L);

        if (SM <= 0.0)
        {
            double rr = ro2 * (SR - u_R) / (SR - SM);
            double e = p2 / g1 + 0.5 * ro2 * (kv(u2) + kv(v2));
            double ee = ((SR - u_R) * e - p2 * u_R + pp * SM) / (SR - SM);
            PQ = SR * (rr * Q_R/ro2 - Q_R) + Q_R * u_R;
            Ps.x = SR * (rr - ro2) + ro2 * u_R;
            Ps.y = SR * (ee - e) + (ga * p2 / (g1) + 0.5 * ro2 * (kv(u2) + kv(v2))) * u_R;

            double F1 = (ro2 * u_R * u_R + p2) + SR * (rr * SM - ro2 * u_R);
            double F2 = (ro2 * u_R * v_R) + SR * (rr * v_R - ro2 * v_R);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else if (SM >= 0.0)
        {
            double rr = ro1 * (SL - u_L) / (SL - SM);
            double e = p1 / g1 + 0.5 * ro1 * (kv(u1) + kv(v1));
            double ee = ((SL - u_L) * e - p1 * u_L + pp * SM) / (SL - SM);
            PQ = SL * (rr * Q_L / ro1 - Q_L) + Q_L * u_L;
            Ps.x = SL * (rr - ro1) + ro1 * u_L;
            Ps.y = SL * (ee - e) + (ga * p1 / (g1) + 0.5 * ro1 * (kv(u1) + kv(v1))) * u_L;
            double F1 = FL1 + SL * (rr * SM - ro1 * u_L);
            double F2 = FL2 + SL * (rr * v_L - ro1 * v_L);
            Pu.x = F1 * n1 + F2 * t1;
            Pu.y = F1 * n2 + F2 * t2;
            return time;
        }
        else
        {
            printf("ERROR HLLC_KOROLKOV_2d   kod oshibki: 1jdt27453h\n");
            return time;
        }
    }
    return time;
}

__device__ double HLLCQ_Aleksashov(const double2& Ls, const double2& Lu, const double2& Rs, const double2& Ru,//
    const double& LQ, const double& RQ, double n1, double n2, double2& Ps, double2& Pu, double& PQ, double rad)
{
    double n[3];
    n[0] = n1;
    n[1] = n2;
    n[2] = 0.0;
    //int id_bn = 1;
    //int n_state = 1;
    double FR[8], FL[8];
    double UL[8], UZ[8], UR[8];
    double UZL[8], UZR[8];

    double vL[3], vR[3], bL[3], bR[3];
    double vzL[3], vzR[3], bzL[3], bzR[3];
    double qv[3];
    double aco[3][3];

    double wv = 0.0;
    double r1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double w1 = 0.0;
    double p1 = Ls.y;
    double bx1 = 0.0;
    double by1 = 0.0;
    double bz1 = 0.0;
    double Q_L = LQ;


    double r2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double w2 = 0.0;
    double p2 = Rs.y;
    double bx2 = 0.0;
    double by2 = 0.0;
    double bz2 = 0.0;
    double Q_R = RQ;

    double ro = (r2 + r1) / 2.0;
    double ap = (p2 + p1) / 2.0;
    double abx = (bx2 + bx1) / 2.0;
    double aby = (by2 + by1) / 2.0;
    double abz = (bz2 + bz1) / 2.0;


    double bk = abx * n[0] + aby * n[1] + abz * n[2];
    double b2 = kv(abx) + kv(aby) + kv(abz);

    double d = b2 - kv(bk);
    aco[0][0] = n[0];
    aco[1][0] = n[1];
    aco[2][0] = n[2];
    if (d > 0.000000001)
    {
        d = sqrt(d);
        aco[0][1] = (abx - bk * n[0]) / d;
        aco[1][1] = (aby - bk * n[1]) / d;
        aco[2][1] = (abz - bk * n[2]) / d;
        aco[0][2] = (aby * n[2] - abz * n[1]) / d;
        aco[1][2] = (abz * n[0] - abx * n[2]) / d;
        aco[2][2] = (abx * n[1] - aby * n[0]) / d;
    }
    else
    {
        double aix, aiy, aiz;
        if ((fabs(n[0]) < fabs(n[1])) && (fabs(n[0]) < fabs(n[2])))
        {
            aix = 1.0;
            aiy = 0.0;
            aiz = 0.0;
        }
        else if (fabs(n[1]) < fabs(n[2]))
        {
            aix = 0.0;
            aiy = 1.0;
            aiz = 0.0;
        }
        else
        {
            aix = 0.0;
            aiy = 0.0;
            aiz = 1.0;
        }

        double aik = aix * n[0] + aiy * n[1] + aiz * n[2];
        d = sqrt(1.0 - kv(aik));
        aco[0][1] = (aix - aik * n[0]) / d;
        aco[1][1] = (aiy - aik * n[1]) / d;
        aco[2][1] = (aiz - aik * n[2]) / d;
        aco[0][2] = (aiy * n[2] - aiz * n[1]) / d;
        aco[1][2] = (aiz * n[0] - aix * n[2]) / d;
        aco[2][2] = (aix * n[1] - aiy * n[0]) / d;
    }

    for (int i = 0; i < 3; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1 + aco[2][i] * w1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2 + aco[2][i] * w2;
        bL[i] = aco[0][i] * bx1 + aco[1][i] * by1 + aco[2][i] * bz1;
        bR[i] = aco[0][i] * bx2 + aco[1][i] * by2 + aco[2][i] * bz2;
    }

    double aaL = bL[0] / sqrt(r1);
    double b2L = kv(bL[0]) + kv(bL[1]) + kv(bL[2]);
    double b21 = b2L / r1;
    double cL = sqrt(ga * p1 / r1);
    double qp = sqrt(b21 + cL * (cL + 2.0 * aaL));
    double qm = sqrt(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / 2.0;
    double ptL = p1 + b2L / 2.0;

    double aaR = bR[0] / sqrt(r2);
    double b2R = kv(bR[0]) + kv(bR[1]) + kv(bR[2]);
    double b22 = b2R / r2;
    double cR = sqrt(ga * p2 / r2);
    qp = sqrt(b22 + cR * (cR + 2.0 * aaR));
    qm = sqrt(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / 2.0;
    double ptR = p2 + b2R / 2.0;

    double aC = (aaL + aaR) / 2.0;
    double b2o = (b22 + b21) / 2.0;
    double cC = sqrt(ga * ap / ro);
    qp = sqrt(b2o + cC * (cC + 2.0 * aC));
    qm = sqrt(b2o + cC * (cC - 2.0 * aC));
    double cfC = (qp + qm) / 2.0;
    double vC1 = (vL[0] + vR[0]) / 2.0;

    double SL = min((vL[0] - cfL), (vR[0] - cfR));
    double SR = max((vL[0] + cfL), (vR[0] + cfR));

    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);

    if (SR <= SL)
    {
        printf("231\n");
    }

    double SM00 = SM;
    double SR00 = SR;
    double SL00 = SL;
    double SM01, SR01, SL01;
    if ((SM00 >= SR00) || (SM00 <= SL00))
    {
        SL = min((vL[0] - cfL), (vR[0] - cfR));
        SR = max((vL[0] + cfL), (vR[0] + cfR));
        suR = SR - vR[0];
        suL = SL - vL[0];
        SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0]) / (suR * r2 - suL * r1);
        SM01 = SM;
        SR01 = SR;
        SL01 = SL;
        if ((SM01 >= SR01) || (SM01 <= SL01))
        {
            printf("251\n");
        }
    }


    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double upt1 = (kv(u1) + kv(v1) + kv(w1)) / 2.0;
    double sbv1 = u1 * bx1 + v1 * by1 + w1 * bz1;

    double upt2 = (kv(u2) + kv(v2) + kv(w2)) / 2.0;
    double sbv2 = u2 * bx2 + v2 * by2 + w2 * bz2;

    double e1 = p1 / g1 + r1 * upt1 + b2L / 2.0;
    double e2 = p2 / g1 + r2 * upt2 + b2R / 2.0;

    double FL0 = Q_L * vL[0];
    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + ptL - kv(bL[0]);
    FL[2] = r1 * vL[0] * vL[1] - bL[0] * bL[1];
    FL[3] = r1 * vL[0] * vL[2] - bL[0] * bL[2];
    FL[4] = (e1 + ptL) * vL[0] - bL[0] * sbv1;
    FL[5] = 0.0;
    FL[6] = vL[0] * bL[1] - vL[1] * bL[0];
    FL[7] = vL[0] * bL[2] - vL[2] * bL[0];

    double FR0 = Q_R * vR[0];
    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + ptR - kv(bR[0]);
    FR[2] = r2 * vR[0] * vR[1] - bR[0] * bR[1];
    FR[3] = r2 * vR[0] * vR[2] - bR[0] * bR[2];
    FR[4] = (e2 + ptR) * vR[0] - bR[0] * sbv2;
    FR[5] = 0.0;
    FR[6] = vR[0] * bR[1] - vR[1] * bR[0];
    FR[7] = vR[0] * bR[2] - vR[2] * bR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;


    for (int ik = 0; ik < 3; ik++)
    {
        UL[ik + 1] = r1 * vL[ik];
        UL[ik + 5] = bL[ik];
        UR[ik + 1] = r2 * vR[ik];
        UR[ik + 5] = bR[ik];
    }

    for (int ik = 0; ik < 8; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }

    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;
    vzR[0] = SM;
    vzL[0] = SM;
    double ptzR = ptR + r2 * suR * (SM - vR[0]);
    double ptzL = ptL + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;
    bzR[0] = UZ[5];
    bzL[0] = UZ[5];

    vzR[1] = UZ[2] / UZ[0];
    vzR[2] = UZ[3] / UZ[0];
    vzL[1] = vzR[1];
    vzL[2] = vzR[2];

    vzR[1] = vR[1] + UZ[5] * (bR[1] - UZ[6]) / suR / r2;
    vzR[2] = vR[2] + UZ[5] * (bR[2] - UZ[7]) / suR / r2;
    vzL[1] = vL[1] + UZ[5] * (bL[1] - UZ[6]) / suL / r1;
    vzL[2] = vL[2] + UZ[5] * (bL[2] - UZ[7]) / suL / r1;

    bzR[1] = UZ[6];
    bzR[2] = UZ[7];
    bzL[1] = bzR[1];
    bzL[2] = bzR[2];

    double sbvz = (UZ[5] * UZ[1] + UZ[6] * UZ[2] + UZ[7] * UZ[3]) / UZ[0];

    double ezR = e2 * suRm + (ptz * SM - ptR * vR[0] + UZ[5] * (sbv2 - sbvz)) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - ptL * vL[0] + UZ[5] * (sbv1 - sbvz)) / (SL - SM);

    if (fabs(UZ[5]) < 0.000000001)
    {
        vzR[1] = vR[1];
        vzR[2] = vR[2];
        vzL[1] = vL[1];
        vzL[2] = vL[2];
        bzR[1] = bR[1] * suRm;
        bzR[2] = bR[2] * suRm;
        bzL[1] = bL[1] * suLm;
        bzL[2] = bL[2] * suLm;
    }
    UZL[0] = rzL;
    UZL[4] = ezL;
    UZR[0] = rzR;
    UZR[4] = ezR;

    for (int ik = 0; ik < 3; ik++)
    {
        UZL[ik + 1] = vzL[ik] * rzL;
        UZL[ik + 5] = bzL[ik];
        UZR[ik + 1] = vzR[ik] * rzR;
        UZR[ik + 5] = bzR[ik];
    }

    if (SL > wv)
    {
        PQ = FL0;
        Ps.x = FL[0] - wv * UL[0];
        Ps.y = FL[4] - wv * UL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] - wv * UL[ik];
        }
    }
    else if ((SL <= wv) && (SM >= wv))
    {
        PQ = FL0 + SL * (rzL * Q_L/r1 - Q_L);
        Ps.x = FL[0] + SL * (rzL - r1) - wv * UZL[0];
        Ps.y = FL[4] + SL * (ezL - e1) - wv * UZL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
        }
    }
    else if ((SM <= wv) && (SR >= wv))
    {
        PQ = FR0 + SR * (rzR * Q_R/r2 - Q_R);
        Ps.x = FR[0] + SR * (rzR - r2) - wv * UZR[0];
        Ps.y = FR[4] + SR * (ezR - e2) - wv * UZR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
        }
    }
    else if (SR < wv)
    {
        PQ = FR0;
        Ps.x = FR[0] - wv * UR[0];
        Ps.y = FR[4] - wv * UR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + -wv * UR[ik];
        }
    }
    else
    {
        printf("DDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDDD\n");
    }


    Pu.x = aco[0][0] * qv[0] + aco[0][1] * qv[1] + aco[0][2] * qv[2];
    Pu.y = aco[1][0] * qv[0] + aco[1][1] * qv[1] + aco[1][2] * qv[2];

    return time;
}

__device__ double HLLC_Aleksashov_2D(double2& Ls, double2& Lu, double2& Rs, double2& Ru,//
    double n1, double n2, double2& Ps, double2& Pu, double rad)
{
    double n[2];
    n[0] = n1;
    n[1] = n2;
    //int id_bn = 1;
    //int n_state = 1;
    double FR[5], FL[5];
    double UL[5], UZ[5], UR[5];
    double UZL[5], UZR[5];

    double vL[2], vR[2];
    double vzL[2], vzR[2];
    double qv[2];
    double aco[2][2];

    double r1 = Ls.x;
    double u1 = Lu.x;
    double v1 = Lu.y;
    double p1 = Ls.y;


    double r2 = Rs.x;
    double u2 = Ru.x;
    double v2 = Ru.y;
    double p2 = Rs.y;

    double ro = (r2 + r1) / 2.0;
    double ap = (p2 + p1) / 2.0;


    aco[0][0] = n[0];
    aco[1][0] = n[1];

    aco[0][1] = -n[1];
    aco[1][1] = n[0];
    

    for (int i = 0; i < 2; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2;
    }


    double cL = sqrt(ga * p1 / r1);
    double cR = sqrt(ga * p2 / r2);


    double cC = sqrt(ga * ap / ro);

    double vC1 = (vL[0] + vR[0]) / 2.0;

    double SL = min((vL[0] - cL), (vR[0] - cR));
    double SR = max((vL[0] + cL), (vR[0] + cR));

    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - p2 + p1 - suL * r1 * vL[0]) / (suR * r2 - suL * r1);


    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double upt1 = (kv(u1) + kv(v1)) / 2.0;

    double upt2 = (kv(u2) + kv(v2)) / 2.0;

    double e1 = p1 / g1 + r1 * upt1;
    double e2 = p2 / g1 + r2 * upt2;

    FL[0] = r1 * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + p1;
    FL[2] = r1 * vL[0] * vL[1];
    FL[4] = (e1 + p1) * vL[0];

    FR[0] = r2 * vR[0];
    FR[1] = r2 * vR[0] * vR[0] + p2;
    FR[2] = r2 * vR[0] * vR[1];
    FR[4] = (e2 + p2) * vR[0];

    UL[0] = r1;
    UL[4] = e1;
    UR[0] = r2;
    UR[4] = e2;


    for (int ik = 0; ik < 2; ik++)
    {
        UL[ik + 1] = r1 * vL[ik];
        UR[ik + 1] = r2 * vR[ik];
    }

    for (int ik = 0; ik < 5; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }

    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;
    vzR[0] = SM;
    vzL[0] = SM;
    double ptzR = p2 + r2 * suR * (SM - vR[0]);
    double ptzL = p1 + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;


    vzR[1] = vR[1];
    vzL[1] = vL[1];



    double ezR = e2 * suRm + (ptz * SM - p2 * vR[0]) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - p1 * vL[0]) / (SL - SM);

    UZL[0] = rzL;
    UZL[4] = ezL;
    UZR[0] = rzR;
    UZR[4] = ezR;

    for (int ik = 0; ik < 2; ik++)
    {
        UZL[ik + 1] = vzL[ik] * rzL;
        UZR[ik + 1] = vzR[ik] * rzR;
    }

    if (SL > 0.0)
    {
        Ps.x = FL[0];
        Ps.y = FL[4];
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FL[ik];
        }
    }
    else if ((SL <= 0.0) && (SM >= 0.0))
    {
        Ps.x = FL[0] + SL * (rzL - r1);
        Ps.y = FL[4] + SL * (ezL - e1);
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]);
        }
    }
    else if ((SM <= 0.0) && (SR >= 0.0))
    {
        Ps.x = FR[0] + SR * (rzR - r2);
        Ps.y = FR[4] + SR * (ezR - e2);
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]);
        }
    }
    else if (SR < 0.0)
    {
        Ps.x = FR[0];
        Ps.y = FR[4];
        for (int ik = 1; ik < 3; ik++)
        {
            qv[ik - 1] = FR[ik];
        }
    }
    else
    {
        printf("hllc 2d ERROR\n");
    }


    Pu.x = aco[0][0] * qv[0] + aco[0][1] * qv[1];
    Pu.y = aco[1][0] * qv[0] + aco[1][1] * qv[1];

    return time;
}

__device__ double HLLDQ_Korolkov(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, double& PQ, const double& n1, const double& n2, const double& n3, const double& rad, int metod, double x, double y)
{// �� ��������, ���� �������� ����� �� �������
 // ������� ����� ����������� ��������� �� ���� ���������

    double bx_L = Bx_L / spi4;
    double by_L = By_L / spi4;
    double bz_L = Bz_L / spi4;

    double bx_R = Bx_R / spi4;
    double by_R = By_R / spi4;
    double bz_R = Bz_R / spi4;

    double t1 = 0.0;
    double t2 = 0.0;
    double t3 = 0.0;

    double m1 = 0.0;
    double m2 = 0.0;
    double m3 = 0.0;

    if (n1 > 0.1)
    {
        t2 = 1.0;
        m3 = 1.0;
    }
    else if (n2 > 0.1)
    {
        t3 = 1.0;
        m1 = 1.0;
    }
    else if (n3 > 0.1)
    {
        t1 = 1.0;
        m2 = 1.0;
    }
    else if (n1 < -0.1)
    {
        t3 = -1.0;
        m2 = -1.0;
    }
    else if (n2 < -0.1)
    {
        t1 = -1.0;
        m3 = -1.0;
    }
    else if (n3 < -0.1)
    {
        t1 = -1.0;
        m2 = -1.0;
    }
    else
    {
        printf("EROROR 1421  normal_error\n");
    }


    double u1, v1, w1, u2, v2, w2;
    u1 = v1_L * n1 + v2_L * n2 + v3_L * n3;
    v1 = v1_L * t1 + v2_L * t2 + v3_L * t3;
    w1 = v1_L * m1 + v2_L * m2 + v3_L * m3;
    u2 = v1_R * n1 + v2_R * n2 + v3_R * n3;
    v2 = v1_R * t1 + v2_R * t2 + v3_R * t3;
    w2 = v1_R * m1 + v2_R * m2 + v3_R * m3;

    double bn1, bt1, bm1, bn2, bt2, bm2;
    bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
    bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
    bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;
    bn2 = bx_R * n1 + by_R * n2 + bz_R * n3;
    bt2 = bx_R * t1 + by_R * t2 + bz_R * t3;
    bm2 = bx_R * m1 + by_R * m2 + bz_R * m3;

    //cout << " = " << bt2 * bt2 + bm2 * bm2 << endl;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double ca_L = bn1 / sqrtroL;
    double ca_R = bn2 / sqrtroR;
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);

    double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);
    double bb_R = kv(bx_R) + kv(by_R) + kv(bz_R);

    double aL = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;
    double aR = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;

    double uu_L = (kv(v1_L) + kv(v2_L) + kv(v3_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R) + kv(v3_R)) / 2.0;

    double cfL = sqrt((ggg * p_L + bb_L + //
        sqrt(kv(ggg * p_L + bb_L) - 4.0 * ggg * p_L * kv(bn1))) / (2.0 * ro_L));
    double cfR = sqrt((ggg * p_R + bb_R + //
        sqrt(kv(ggg * p_R + bb_R) - 4.0 * ggg * p_R * kv(bn2))) / (2.0 * ro_R));


    double SL = min(u1, u2) - max(cfL, cfR);
    double SR = max(u1, u2) + max(cfL, cfR);

    double pTL = p_L + bb_L / 2.0;
    double pTR = p_R + bb_R / 2.0;

    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - pTR + pTL) //
        / (suR * ro_R - suL * ro_L);

    double PTT = (suR * ro_R * pTL - suL * ro_L * pTR + ro_L * ro_R * suR * suL * (u2 - u1))//
        / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double FL[9], FR[9], UL[9], UR[9];

    double e1 = p_L / g1 + ro_L * uu_L + bb_L / 2.0;
    double e2 = p_R / g1 + ro_R * uu_R + bb_R / 2.0;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + pTL - kv(bn1);
    FL[2] = ro_L * u1 * v1 - bn1 * bt1;
    FL[3] = ro_L * u1 * w1 - bn1 * bm1;
    FL[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
    //cout << uu_L << endl;
    FL[5] = 0.0;
    FL[6] = u1 * bt1 - v1 * bn1;
    FL[7] = u1 * bm1 - w1 * bn1;
    FL[8] = Q_L * u1;

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + pTR - kv(bn2);
    FR[2] = ro_R * u2 * v2 - bn2 * bt2;
    FR[3] = ro_R * u2 * w2 - bn2 * bm2;
    FR[4] = (e2 + pTR) * u2 - bn2 * (u2 * bn2 + v2 * bt2 + w2 * bm2);
    FR[5] = 0.0;
    FR[6] = u2 * bt2 - v2 * bn2;
    FR[7] = u2 * bm2 - w2 * bn2;
    FR[8] = Q_R * u2;

    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = ro_L * w1;
    UL[4] = e1;
    UL[5] = bn1;
    UL[6] = bt1;
    UL[7] = bm1;
    UL[8] = Q_L;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = ro_R * w2;
    UR[4] = e2;
    UR[5] = bn2;
    UR[6] = bt2;
    UR[7] = bm2;
    UR[8] = Q_R;

    double bn = (SR * UR[5] - SL * UL[5] + FL[5] - FR[5]) / (SR - SL);
    double bt = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
    double bm = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
    double bbn = bn * bn;

    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);

    if (metod == 2)   // HLLC  + mgd
    {
        double sbv1 = u1 * bn1 + v1 * bt1 + w1 * bm1;
        double sbv2 = u2 * bn2 + v2 * bt2 + w2 * bm2;

        double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
        double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
        double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
        double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
        double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
        double vzL, vzR, vLL, wLL, vRR, wRR, ppLR, btt1, bmm1, btt2, bmm2, ee1, ee2;


        double suRm = suR / (SR - SM);
        double suLm = suL / (SL - SM);
        double rzR = ro_R * suRm;
        double rzL = ro_L * suLm;

        double ptzR = pTR + ro_R * suR * (SM - u2);
        double ptzL = pTL + ro_L * suL * (SM - u1);
        double ptz = (ptzR + ptzL) / 2.0;


        vRR = UZ2 / UZ0;                 // ������������!!!!
        wRR = UZ3 / UZ0;
        vLL = vRR;
        wLL = wRR;

        //vRR = v2 + bn * (bt2 - bt) / suR / ro_R;  // �� ������������!!!
        //wRR = w2 + bn * (bm2 - bm) / suR / ro_R;
        //vLL = v1 + bn * (bt1 - bt) / suL / ro_L;
        //wLL = w1 + bn * (bm1 - bm) / suL / ro_L;


        btt2 = bt;
        bmm2 = bm;
        btt1 = btt2;
        bmm1 = bmm2;

        double sbvz = (bn * UZ1 + bt * UZ2 + bm * UZ3) / UZ0;

        ee2 = e2 * suRm + (ptz * SM - pTR * u2 + bn * (sbv2 - sbvz)) / (SR - SM);
        ee1 = e1 * suLm + (ptz * SM - pTL * u1 + bn * (sbv1 - sbvz)) / (SL - SM);

        //if (fabs(bn) < 0.000001 ) // ���� �������������
        //{
        //    vRR = v2;
        //    wRR = w2;
        //    vLL = v1;
        //    wLL = w1;
        //    btt2 = bt2 * suRm;
        //    bmm2 = bm2 * suRm;
        //    btt1 = bt1 * suLm;
        //    bmm1 = bm1 * suLm;
        //}

        /*ppLR = (pTL + ro_L * (SL - u1) * (SM - u1) + pTR + ro_R * (SR - u2) * (SM - u2)) / 2.0;

        if (fabs(bn) < 0.000001)
        {
            vLL = v1;
            wLL = w1;
            vRR = v2;
            wRR = w2;

            btt1 = bt1 * (SL - u1) / (SL - SM);
            btt2 = bt2 * (SR - u2) / (SR - SM);

            bmm1 = bm1 * (SL - u1) / (SL - SM);
            bmm2 = bm2 * (SR - u2) / (SR - SM);

            ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM) / (SL - SM);
            ee2 = ((SR - u2) * e2 - pTL * u2 + ppLR * SM) / (SR - SM);
        }
        else
        {
            btt2 = btt1 = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
            bmm2 = bmm1 = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
            vLL = v1 + bn * (bt1 - btt1) / (ro_L * (SL - u1));
            vRR = v2 + bn * (bt2 - btt2) / (ro_R * (SR - u2));

            wLL = w1 + bn * (bm1 - bmm1) / (ro_L * (SL - u1));
            wRR = w2 + bn * (bm2 - bmm2) / (ro_R * (SR - u2));

            double sks1 = u1 * bn1 + v1 * bt1 + w1 * bm1 - SM * bn - vLL * btt1 - wLL * bmm1;
            double sks2 = u2 * bn2 + v2 * bt2 + w2 * bm2 - SM * bn - vRR * btt2 - wRR * bmm2;

            ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM + bn * sks1) / (SL - SM);
            ee2 = ((SR - u2) * e2 - pTR * u2 + ppLR * SM + bn * sks2) / (SR - SM);
        }*/


        double  ULL[9], URR[9], PO[9];
        ULL[0] = ro_LL;
        ULL[1] = ro_LL * SM;
        ULL[2] = ro_LL * vLL;
        ULL[3] = ro_LL * wLL;
        ULL[4] = ee1;
        ULL[5] = bn;
        ULL[6] = btt1;
        ULL[7] = bmm1;
        ULL[8] = Q_LL;

        URR[0] = ro_RR;
        URR[1] = ro_RR * SM;
        URR[2] = ro_RR * vRR;
        URR[3] = ro_RR * wRR;
        URR[4] = ee2;
        URR[5] = bn;
        URR[6] = btt2;
        URR[7] = bmm2;
        URR[8] = Q_RR;

        if (SL >= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i];
            }
        }
        else if (SL < 0.0 && SM >= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
            }
        }
        else if (SR > 0.0 && SM < 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
            }
        }
        else if (SR <= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i];
            }
        }



        double SN = max(fabs(SL), fabs(SR));

        PO[5] = -SN * (bn2 - bn1);

        P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
        P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
        P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
        P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
        P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
        P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
        P[0] = PO[0];
        P[4] = PO[4];
        PQ = PO[8];

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;

    }
    else if (metod == 3)  // HLLD
    {

        double ttL = ro_L * suL * (SL - SM) - bbn;
        double ttR = ro_R * suR * (SR - SM) - bbn;

        double vLL, wLL, vRR, wRR, btt1, bmm1, btt2, bmm2;

        if (fabs(ttL) >= 0.00001)
        {
            vLL = v1 - bn * bt1 * (SM - u1) / ttL;
            wLL = w1 - bn * bm1 * (SM - u1) / ttL;
            btt1 = bt1 * (ro_L * suL * suL - bbn) / ttL;
            bmm1 = bm1 * (ro_L * suL * suL - bbn) / ttL;
        }
        else
        {
            //printf("ttl = 0   kod:1319, %lf, %lf, %lf, %lf\n", x, y, (SL - SM), bbn);
            vLL = v1;
            wLL = w1;
            btt1 = 0.0;
            bmm1 = 0.0;
        }

        if (fabs(ttR) >= 0.00001)
        {
            vRR = v2 - bn * bt2 * (SM - u2) / ttR;
            wRR = w2 - bn * bm2 * (SM - u2) / ttR;
            btt2 = bt2 * (ro_R * suR * suR - bbn) / ttR;
            bmm2 = bm2 * (ro_R * suR * suR - bbn) / ttR;
            //cout << "tbr = " << (ro_R * suR * suR - bbn) / ttR << endl;
            //cout << "bt2 = " << bt2 << endl;
        }
        else
        {
            //printf("ttR = 0   kod:1337, %lf, %lf, %lf\n", x, y, ttR);
            vRR = v2;
            wRR = w2;
            btt2 = 0.0;
            bmm2 = 0.0;
        }

        double eLL = (e1 * suL + PTT * SM - pTL * u1 + bn * //
            ((u1 * bn1 + v1 * bt1 + w1 * bm1) - (SM * bn + vLL * btt1 + wLL * bmm1))) //
            / (SL - SM);
        double eRR = (e2 * suR + PTT * SM - pTR * u2 + bn * //
            ((u2 * bn2 + v2 * bt2 + w2 * bm2) - (SM * bn + vRR * btt2 + wRR * bmm2))) //
            / (SR - SM);

        double sqrtroLL = sqrt(ro_LL);
        double sqrtroRR = sqrt(ro_RR);
        double SLL = SM - fabs(bn) / sqrtroLL;
        double SRR = SM + fabs(bn) / sqrtroRR;

        double idbn = 1.0;
        if (fabs(bn) > 0.000001)
        {
            //printf("not idbn = 0   kod:1359 \n");
            idbn = 1.0 * sign_(bn);
        }
        else
        {
            //printf("idbn = 0   kod:1363 \n");
            idbn = 0.0;
            SLL = SM;
            SRR = SM;
        }

        double vLLL = (sqrtroLL * vLL + sqrtroRR * vRR + //
            idbn * (btt2 - btt1)) / (sqrtroLL + sqrtroRR);

        double wLLL = (sqrtroLL * wLL + sqrtroRR * wRR + //
            idbn * (bmm2 - bmm1)) / (sqrtroLL + sqrtroRR);

        double bttt = (sqrtroLL * btt2 + sqrtroRR * btt1 + //
            idbn * sqrtroLL * sqrtroRR * (vRR - vLL)) / (sqrtroLL + sqrtroRR);

        double bmmm = (sqrtroLL * bmm2 + sqrtroRR * bmm1 + //
            idbn * sqrtroLL * sqrtroRR * (wRR - wLL)) / (sqrtroLL + sqrtroRR);

        double eLLL = eLL - idbn * sqrtroLL * ((SM * bn + vLL * btt1 + wLL * bmm1) //
            - (SM * bn + vLLL * bttt + wLLL * bmmm));
        double eRRR = eRR + idbn * sqrtroRR * ((SM * bn + vRR * btt2 + wRR * bmm2) //
            - (SM * bn + vLLL * bttt + wLLL * bmmm));
        //cout << " = " << bn << " " << btt2 << " " << bmm2 << endl;
        //cout << "sbvr = " << (SM * bn + vRR * btt2 + wRR * bmm2) << endl;
        double  ULL[9], URR[9], ULLL[9], URRR[9];

        ULL[0] = ro_LL;
        ULL[1] = ro_LL * SM;
        ULL[2] = ro_LL * vLL;
        ULL[3] = ro_LL * wLL;
        ULL[4] = eLL;
        ULL[5] = bn;
        ULL[6] = btt1;
        ULL[7] = bmm1;
        ULL[8] = Q_LL;

        URR[0] = ro_RR;
        //cout << ro_RR << endl;
        URR[1] = ro_RR * SM;
        URR[2] = ro_RR * vRR;
        URR[3] = ro_RR * wRR;
        URR[4] = eRR;
        URR[5] = bn;
        URR[6] = btt2;
        URR[7] = bmm2;
        URR[8] = Q_RR;

        ULLL[0] = ro_LL;
        ULLL[1] = ro_LL * SM;
        ULLL[2] = ro_LL * vLLL;
        ULLL[3] = ro_LL * wLLL;
        ULLL[4] = eLLL;
        ULLL[5] = bn;
        ULLL[6] = bttt;
        ULLL[7] = bmmm;
        ULLL[8] = Q_LL;

        URRR[0] = ro_RR;
        URRR[1] = ro_RR * SM;
        URRR[2] = ro_RR * vLLL;
        URRR[3] = ro_RR * wLLL;
        URRR[4] = eRRR;
        URRR[5] = bn;
        URRR[6] = bttt;
        URRR[7] = bmmm;
        URRR[8] = Q_RR;

        double PO[9];

        if (SL >= 0.0)
        {
            //cout << "SL >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i];
            }
        }
        else if (SL < 0.0 && SLL >= 0.0)
        {
            //cout << "SL < 0.0 && SLL >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
            }
            //cout << ULL[0] << endl;
        }
        else if (SLL <= 0.0 && SM >= 0.0)
        {
            //cout << "SLL <= 0.0 && SM >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SLL * ULLL[i] - (SLL - SL) * ULL[i] - SL * UL[i];
            }
        }
        else if (SM < 0.0 && SRR > 0.0)
        {
            //cout << "SM < 0.0 && SRR > 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SRR * URRR[i] - (SRR - SR) * URR[i] - SR * UR[i];
            }
            //cout << "P4 = " << URRR[4] << endl;
        }
        else if (SR > 0.0 && SRR <= 0.0)
        {
            //cout << "SR > 0.0 && SRR <= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
            }
            //cout << URR[0] << endl;
        }
        else if (SR <= 0.0)
        {
            //cout << "SR <= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i];
            }
        }



        double SN = max(fabs(SL), fabs(SR));

        PO[5] = -SN * (bn2 - bn1);

        P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
        P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
        P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
        P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
        P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
        P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
        P[0] = PO[0];
        P[4] = PO[4];
        PQ = PO[8];

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }

}


__device__ double HLLDQ_Korolkov2(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, double& PQ, const double& n1, const double& n2, const double& n3, const double& rad, int metod, double x, double y)
{// �� ��������, ���� �������� ����� �� �������
 // ������� ����� ����������� ��������� �� ���� ���������

    /*dimension qqq(8), qqq1(8), qqq2(8)
        dimension FR(8), FL(8)
        dimension FW(8), UL(8), UZ(8), UR(8)
        dimension UZL(8), UZR(8)
        dimension UZZL(8), UZZR(8)
        dimension dq(8)

        dimension vL(3), vR(3), bL(3), bR(3)
        dimension vzL(3), vzR(3), bzL(3), bzR(3)
        dimension vzzL(3), vzzR(3), bzzL(3), bzzR(3)
        dimension aco(3, 3), qv(3), qb(3)*/

    double aco[3][3];
    double vL[3], vR[3], bL[3], bR[3], FL[9], FR[9], UL[9], UR[9], UZ[9], vzL[3], vzR[3], bzL[3], bzR[3], UZL[9], UZR[9], qqq[9];

    double eps = 1E-12;
    double epsb = 1E-06;
    double eps_p = 1E-06;
    double eps_d = 1E-03;



    double wv = 0.0;


    double r1 = ro_L;
    double u1 = v1_L;
    double v1 = v2_L;
    double w1 = v3_L;
    double p1 = p_L;
    double bx1 = Bx_L / spi4;
    double by1 = By_L / spi4;
    double bz1 = Bz_L / spi4;


    double r2 = ro_R;
    double u2 = v1_R;
    double v2 = v2_R;
    double w2 = v3_R;
    double p2 = p_R;
    double bx2 = Bx_R / spi4;
    double by2 = By_R / spi4;
    double bz2 = Bz_R / spi4;

    double ro = (r2 + r1) / 2.0;
    double au = (u2 + u1) / 2.0;
    double av = (v2 + v1) / 2.0;
    double aw = (w2 + w1) / 2.0;
    double ap = (p2 + p1) / 2.0;
    double abx = (bx2 + bx1) / 2.0;
    double aby = (by2 + by1) / 2.0;
    double abz = (bz2 + bz1) / 2.0;


    double bk = abx * n1 + aby * n2 + abz * n3;
    double b2 = kv(abx) + kv(aby) + kv(abz);

    double d = b2 - kv(bk);
    aco[0][0] = n1;
    aco[1][0] = n2;
    aco[2][0] = n3;

    double aix, aiy, aiz, aik;

    if (d > eps)
    {
        d = sqrt(d);
        aco[0][1] = (abx - bk * n1) / d;
        aco[1][1] = (aby - bk * n2) / d;
        aco[2][1] = (abz - bk * n3) / d;
        aco[0][2] = (aby * n3 - abz * n2) / d;
        aco[1][2] = (abz * n1 - abx * n3) / d;
        aco[2][2] = (abx * n2 - aby * n1) / d;
    }
    else
    {
        if (fabs(n1) < fabs(n2) && fabs(n1) < fabs(n3))
        {
            aix = 1.0;
            aiy = 0.0;
            aiz = 0.0;
        }
        else if (fabs(n2) < fabs(n3))
        {
            aix = 0.0;
            aiy = 1.0;
            aiz = 0.0;
        }
        else
        {
            aix = 0.0;
            aiy = 0.0;
            aiz = 1.0;
        }
        aik = aix * n1 + aiy * n2 + aiz * n3;
        d = sqrt(1.0 - kv(aik));
        aco[0][1] = (aix - aik * n1) / d;
        aco[1][1] = (aiy - aik * n2) / d;
        aco[2][1] = (aiz - aik * n3) / d;
        aco[0][2] = (aiy * n3 - aiz * n2) / d;
        aco[1][2] = (aiz * n1 - aix * n3) / d;
        aco[2][2] = (aix * n2 - aiy * n1) / d;
    }

    for (int i = 0; i < 3; i++)
    {
        vL[i] = aco[0][i] * u1 + aco[1][i] * v1 + aco[2][i] * w1;
        vR[i] = aco[0][i] * u2 + aco[1][i] * v2 + aco[2][i] * w2;
        bL[i] = aco[0][i] * bx1 + aco[1][i] * by1 + aco[2][i] * bz1;
        bR[i] = aco[0][i] * bx2 + aco[1][i] * by2 + aco[2][i] * bz2;
    }

    double aaL = bL[0] / sqrt(r1);
    double b2L = kv(bL[0]) + kv(bL[1]) + kv(bL[2]);
    double b21 = b2L / r1;
    double cL = sqrt(ga * p1 / r1);
    double qp = sqrt(b21 + cL * (cL + 2.0 * aaL));
    double qm = sqrt(b21 + cL * (cL - 2.0 * aaL));
    double cfL = (qp + qm) / 2.0;
    double ptL = p1 + b2L / 2.0;

    double aaR = bR[0] / sqrt(r2);
    double b2R = kv(bR[0]) + kv(bR[1]) + kv(bR[2]);
    double b22 = b2R / r2;
    double cR = sqrt(ga * p2 / r2);
    qp = sqrt(b22 + cR * (cR + 2.0 * aaR));
    qm = sqrt(b22 + cR * (cR - 2.0 * aaR));
    double cfR = (qp + qm) / 2.0;
    double ptR = p2 + b2R / 2.0;

    double aC = (aaL + aaR) / 2.0;
    double b2o = (b22 + b21) / 2.0;
    double cC = sqrt(ga * ap / ro);
    qp = sqrt(b2o + cC * (cC + 2.0 * aC));
    qm = sqrt(b2o + cC * (cC - 2.0 * aC));
    double cfC = (qp + qm) / 2.0;
    double vC1 = (vL[0] + vR[0]) / 2.0;

    double SL = min((vL[0] - cfL), (vC1 - cfC));
    double SR = max((vR[0] + cfR), (vC1 + cfC));

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double suR = SR - vR[0];
    double suL = SL - vL[0];
    double SM = (suR * r2 * vR[0] - ptR + ptL - suL * r1 * vL[0])
        / (suR * r2 - suL * r1);



    double upt1 = (kv(u1) + kv(v1) + kv(w1)) / 2.0;
    double sbv1 = u1 * bx1 + v1 * by1 + w1 * bz1;

    double upt2 = (kv(u2) + kv(v2) + kv(w2)) / 2.0;
    double sbv2 = u2 * bx2 + v2 * by2 + w2 * bz2;

    double e1 = p1 / g1 + r1 * upt1 + b2L / 2.0;
    double e2 = p2 / g1 + r2 * upt2 + b2R / 2.0;

    FL[0] = r1 * vL[0];
    FL[8] = Q_L * vL[0];
    FL[1] = r1 * vL[0] * vL[0] + ptL - kv(bL[0]);
    FL[2] = r1 * vL[0] * vL[1] - bL[0] * bL[1];
    FL[3] = r1 * vL[0] * vL[2] - bL[0] * bL[2];
    FL[4] = (e1 + ptL) * vL[0] - bL[0] * sbv1;
    FL[5] = 0.0;
    FL[6] = vL[0] * bL[1] - vL[1] * bL[0];
    FL[7] = vL[0] * bL[2] - vL[2] * bL[0];

    FR[0] = r2 * vR[0];
    FR[8] = Q_R * vL[0];
    FR[1] = r2 * vR[0] * vR[0] + ptR - kv(bR[0]);
    FR[2] = r2 * vR[0] * vR[1] - bR[0] * bR[1];
    FR[3] = r2 * vR[0] * vR[2] - bR[0] * bR[2];
    FR[4] = (e2 + ptR) * vR[0] - bR[0] * sbv2;
    FR[5] = 0.0;
    FR[6] = vR[0] * bR[1] - vR[1] * bR[0];
    FR[7] = vR[0] * bR[2] - vR[2] * bR[0];

    UL[0] = r1;
    UL[8] = Q_L;
    UL[4] = e1;
    UR[0] = r2;
    UR[8] = Q_R;
    UR[4] = e2;

    for (int ik = 0; ik < 3; ik++)
    {
        UL[ik + 1] = r1 * vL[ik];
        UL[ik + 5] = bL[ik];
        UR[ik + 1] = r2 * vR[ik];
        UR[ik + 5] = bR[ik];
    }

    for (int ik = 0; ik < 9; ik++)
    {
        UZ[ik] = (SR * UR[ik] - SL * UL[ik] + FL[ik] - FR[ik]) / (SR - SL);
    }


    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double QzR = Q_R * suRm;
    double QzL = Q_L * suLm;
    double rzR = r2 * suRm;
    double rzL = r1 * suLm;
    vzR[0] = SM;
    vzL[0] = SM;
    double ptzR = ptR + r2 * suR * (SM - vR[0]);
    double ptzL = ptL + r1 * suL * (SM - vL[0]);
    double ptz = (ptzR + ptzL) / 2.0;
    bzR[0] = UZ[5];
    bzL[0] = UZ[5];

    vzR[1] = UZ[2] / UZ[0];
    vzR[2] = UZ[3] / UZ[0];
    vzL[1] = vzR[1];
    vzL[2] = vzR[2];

    vzR[1] = vR[1] + UZ[5] * (bR[1] - UZ[6]) / suR / r2;
    vzR[2] = vR[2] + UZ[5] * (bR[2] - UZ[7]) / suR / r2;
    vzL[1] = vL[1] + UZ[5] * (bL[1] - UZ[6]) / suL / r1;
    vzL[2] = vL[2] + UZ[5] * (bL[2] - UZ[7]) / suL / r1;

    bzR[1] = UZ[6];
    bzR[2] = UZ[7];
    bzL[1] = bzR[1];
    bzL[2] = bzR[2];

    double sbvz = (UZ[5] * UZ[1] + UZ[6] * UZ[2] + UZ[7] * UZ[3]) / UZ[0];

    double ezR = e2 * suRm + (ptz * SM - ptR * vR[0] + UZ[5] * (sbv2 - sbvz)) / (SR - SM);
    double ezL = e1 * suLm + (ptz * SM - ptL * vL[0] + UZ[5] * (sbv1 - sbvz)) / (SL - SM);

    if (fabs(UZ[5]) < epsb)
    {
        vzR[1] = vR[1];
        vzR[2] = vR[2];
        vzL[1] = vL[1];
        vzL[2] = vL[2];
        bzR[1] = bR[1] * suRm;
        bzR[2] = bR[2] * suRm;
        bzL[1] = bL[1] * suLm;
        bzL[2] = bL[2] * suLm;
    }
    UZL[0] = rzL;
    UZL[8] = QzL;
    UZL[4] = ezL;
    UZR[0] = rzR;
    UZR[8] = QzR;
    UZR[4] = ezR;
    
    for (int ik = 0; ik < 3; ik++)
    {
        UZL[ik + 1] = vzL[ik] * rzL;
        UZL[ik + 5] = bzL[ik];
        UZR[ik + 1] = vzR[ik] * rzR;
        UZR[ik + 5] = bzR[ik];
    }

    double qv[3], qb[3];

    if (SL > wv)
    {
        qqq[0] = FL[0] - wv * UL[0];
        qqq[8] = FL[8] - wv * UL[8];
        qqq[4] = FL[4] - wv * UL[4];

        for (int ik = 1; ik < 4; ik++) 
        {
            qv[ik - 1] = FL[ik] - wv * UL[ik];
        }
        for (int ik = 5; ik < 8; ik++)
        {
            qb[ik - 5] = FL[ik] - wv * UL[ik];
        }
    }

    if (SL <= wv && SM >= wv)
    {
        qqq[0] = FL[0] + SL * (rzL - r1) - wv * UZL[0];
        qqq[8] = FL[8] + SL * (QzL - Q_L) - wv * UZL[8];
        qqq[4] = FL[4] + SL * (ezL - e1) - wv * UZL[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
        }
        for (int ik = 5; ik < 8; ik++)
        {
            qb[ik - 5] = FL[ik] + SL * (UZL[ik] - UL[ik]) - wv * UZL[ik];
        }
    }

    if (SM <= wv && SR >= wv)
    {
        qqq[0] = FR[0] + SR * (rzR - r2) - wv * UZR[0];
        qqq[8] = FR[8] + SR * (QzR - Q_R) - wv * UZR[8];
        qqq[4] = FR[4] + SR * (ezR - e2) - wv * UZR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
        }
        for (int ik = 5; ik < 8; ik++)
        {
            qb[ik - 5] = FR[ik] + SR * (UZR[ik] - UR[ik]) - wv * UZR[ik];
        }
    }

    if (SR < wv)
    {
        qqq[0] = FR[0] - wv * UR[0];
        qqq[8] = FR[8] - wv * UR[8];
        qqq[4] = FR[4] - wv * UR[4];
        for (int ik = 1; ik < 4; ik++)
        {
            qv[ik - 1] = FR[ik] - wv * UR[ik];
        }
        for (int ik = 5; ik < 8; ik++)
        {
            qb[ik - 5] = FR[ik] - wv * UR[ik];
        }
    }

    //double SN = max(fabs(SL), fabs(SR));
    //qb[0] = -SN * (bR[0] - bL[0]);

    for (int i = 0; i < 3; i++)
    {
        qqq[i + 1] = aco[i][0] * qv[0] + aco[i][1] * qv[1] + aco[i][2] * qv[2];
        qqq[i + 5] = aco[i][0] * qb[0] + aco[i][1] * qb[1] + aco[i][2] * qb[2];
        qqq[i + 5] = spi4 * qqq[i + 5];
    }



    P[0] = qqq[0];
    P[1] = qqq[1];
    P[2] = qqq[2];
    P[3] = qqq[3];
    P[4] = qqq[5];
    P[5] = qqq[6];
    P[6] = qqq[7];
    P[7] = qqq[4];
    PQ = qqq[8];
    return time;

    

    
}
